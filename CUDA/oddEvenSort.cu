
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 10 
#define BLOCKS 1
#define THREADS_PER_BLOCK 5

__global__ void OddEvensort(int *array, int size) {

	bool odd = true;
	__shared__ bool swappedodd;
	__shared__ bool swappedeven;

	int temp;
	swappedodd  = true;
	swappedeven = true;
	
	while (true) {
		if (odd == true) {
			//Swapping at Odd locations
			__syncthreads();
			swappedodd = false;
			__syncthreads();
			
			int idx = threadIdx.x + blockIdx.x * blockDim.x;
			
			if (idx<(size / 2)) {
				if (array[2 * idx]>array[2 * idx + 1]) {
					printf("\nThread Id %d : is swapping %d <-> %d \
					\nThread Id %d : [%d] <-> [%d]\n", idx, array[2 * idx], \
					array[2 * idx + 1], idx, 2 * idx, (2 * idx + 1));
					//swap the numbers at odd location (array[2*idx],array[2*idx+1]);
					temp = array[2 * idx];
					array[2 * idx] = array[2 * idx + 1];
					array[2 * idx + 1] = temp;
					swappedodd = true;
				}
			}
			__syncthreads();
		
		} else {
			//Swapping at Even locations
			__syncthreads();
			swappedeven = false;
			__syncthreads();

			int idx = threadIdx.x + blockIdx.x * blockDim.x;
			if (idx<(size / 2) - 1) {
				if (array[2 * idx + 1] > array[2 * idx + 2]) {
					printf("\nThread Id %d : is swapping %d <-> %d\
					\nThread Id %d : [%d] <-> [%d]\n\
					", idx, array[2 * idx + 1], array[2 * idx + 2], idx, \
					(2 * idx + 1), (2 * idx + 2));
					//swap the numbers at even location(array[2*idx+1],array[2*idx+2]);
					temp = array[2 * idx + 1];
					array[2 * idx + 1] = array[2 * idx + 2];
					array[2 * idx + 2] = temp;
					swappedeven = true;
				}
			}
			__syncthreads();
		}

		if (!(swappedodd || swappedeven))
			break;

		odd = !odd;
	}
	__syncthreads();
}

int main(void)
{
	int *a, *a_sorted, i;
	int *d_a;
	int size = sizeof(int)*SIZE;

	hipMalloc((void**)&d_a, size);

	a = (int*)malloc(size);
	a_sorted = (int*)malloc(size);

	printf("\nEnter %d numbers to sort: ", SIZE);
	for (i = 0; i< SIZE; i++) {
		scanf("%d", &a[i]);
	}

	printf("\nUnsorted array is:\n");
	for (i = 0; i < SIZE; i++) {
		printf("%d ", a[i]);
	}printf("\n");

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

	OddEvensort<<<BLOCKS, THREADS_PER_BLOCK>>>(d_a, SIZE);

	hipMemcpy(a_sorted, d_a, size, hipMemcpyDeviceToHost);
	
	printf("\nSorted array is:\n");
	for (i = 0; i<SIZE; i++) {
		printf("%d ", a_sorted[i]);
	}printf("\n");

	free(a);
	free(a_sorted);
	hipFree(d_a);
	
	return 0;
}
