
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define NUM_BLOCKS 5
#define THREADS_PER_BLOCK 20
#define SIZE 100

__global__ void DotProd(int *a, int *b, int *c) {
	__shared__ int temp[THREADS_PER_BLOCK];

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	printf("Block ID: %d\tThread ID: %d\n", blockIdx.x, threadIdx.x);
	temp[threadIdx.x] = a[x] * b[x];
	printf("Temp %d: %d\n", x, temp[threadIdx.x]);

	__syncthreads();
	
	if (threadIdx.x == 0) 
	{
		int i,sum = 0;
		for (i = 0; i < THREADS_PER_BLOCK; i++) 
		{
			sum += temp[i];
		}
		printf("\nSUM[%d]:%d", blockIdx.x, sum);
		atomicAdd(c, sum);	
	}
}

int main() {
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int n = SIZE * sizeof(int);
	int i;
	
	//1 : Allocate memory for Host and Device variables
	a = (int*)malloc(n);
	b = (int*)malloc(n);
	c = (int*)malloc(sizeof(int));
	
	hipMalloc((void**)&d_a, n);
	hipMalloc((void**)&d_b, n);
	hipMalloc((void**)&d_c, sizeof(int));

	//2: Initialize Host variables
	*c = 0;
	for (i = 0; i < SIZE; i++) {
		a[i] = i + 1;
		b[i] = i + 1;
	}

	// Display the values of the arrays
	printf("\nArray A:\n");
	for (i = 0; i < SIZE; i++) {
		printf("%d ", a[i]);
	}
	printf("\nArray B:\n");
	for (i = 0; i < SIZE; i++) {
			printf("%d ", b[i]);
	}
	printf("\n");
	//3: Copy data to device variables.
	hipMemcpy(d_a, a, n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, sizeof(int), hipMemcpyHostToDevice);

	//4: Launch the Kernel
	printf("\nLaunching Kernel\n");
	DotProd <<<NUM_BLOCKS, THREADS_PER_BLOCK>>> (d_a, d_b, d_c);

	//5: Copy results from device to Host.
	hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("\nDot Product is: %d\n", *c);
	
	//6: Free Memory
	free(a); free(b); free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return 0;
}


