//Matrix Multiplication

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 2
#define GRID_SIZE 2
#define N GRID_SIZE * BLOCK_SIZE

__global__ void MatrixMul(float *A, float *B, float *C, int n)
{
	// Each thread computes a single element of C
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	float sum = 0;
	for (int i = 0; i < n; ++i) {
		sum += (A[row*n + i] * B[i*n + col]);
	}
		
	C[row*n + col] = sum;
	printf("\n Block[%d][%d] : Thread[%d][%d] : Product = %.2f\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, sum);
}

int main()
{
	// Perform matrix multiplication C = A*B
	// where A, B and C are NxN matrices
	// Restricted to matrices where N = GRID_SIZE*BLOCK_SIZE;
	float *hA, *hB, *hC;
	float *dA, *dB, *dC;
	int size = N * N * sizeof(float);

	printf("Matrix Multiplcation:-->\n");
	printf("Matrix size: %d x %d\n", N,N);

	// Allocate memory on the host
	hA = (float *) malloc(size);
	hB = (float *) malloc(size);
	hC = (float *) malloc(size);

	// Initialize matrices on the host
	for (int j = 0; j<N; j++){
		for (int i = 0; i<N; i++){
			hA[j*N + i] = 3;
			hB[j*N + i] = 2;
		}
	}

	printf("Matrix 1:\n");
	for (int j = 0; j<N; j++){
		for (int i = 0; i<N; i++){
			printf("%.2f ", hA[j*N + i]);
		}
		printf("\n");
	}
	
	printf("\nMatrix 2:\n");
	for (int j = 0; j<N; j++){
		for (int i = 0; i<N; i++){
			printf("%.2f ", hB[j*N + i]);
		}
		printf("\n");
	}
	// Allocate memory on the device
	hipMalloc(&dA, size);
	hipMalloc(&dB, size);
	hipMalloc(&dC, size);

	dim3 threadBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(GRID_SIZE, GRID_SIZE);

	// Copy matrices from the host to device
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

	//Execute the matrix multiplication kernel
	printf("\n Kernel Launch with Gird of size (%dx%d) and Block of size (%dx%d)\n", GRID_SIZE, GRID_SIZE, BLOCK_SIZE, BLOCK_SIZE);
	MatrixMul <<<grid, threadBlock >>>(dA, dB, dC, N);

	// Now copy the GPU result back to CPU
	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

	printf("\nThe Product of Matrix A and B is:\n");
	for (int j = 0; j<N; j++){
		for (int i = 0; i<N; i++){
			printf("%.2f ", hC[j*N + i]);
		}
		printf("\n");
	}
	return 0;
}
